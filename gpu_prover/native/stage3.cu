#include "hip/hip_runtime.h"
#include "arg_utils.cuh"
#include "context.cuh"
#include "ops_complex.cuh"
#include "vectorized.cuh"

using namespace field;
using namespace memory;

using bf = base_field;
using e2 = ext2_field;
using e4 = ext4_field;

/// These values are hand-picked, so that the biggest circuit (bigint) fits.
/// What is here must match values from stage_3_kernels.rs
constexpr unsigned MAX_NON_BOOLEAN_CONSTRAINTS = 192;
constexpr unsigned MAX_TERMS = 1824;
constexpr unsigned MAX_EXPLICIT_COEFFS = 632;
constexpr unsigned MAX_FLAT_COL_IDXS = 3488;
constexpr uint8_t COEFF_IS_ONE = 0x00;
constexpr uint8_t COEFF_IS_MINUS_ONE = 0x01;
// constexpr uint8_t COEFF_IS_EXPLICIT = 0x02; // technically unused, "default" case

extern "C" struct FlattenedGenericConstraintsMetadata {
  const uint8_t coeffs_info[MAX_TERMS];
  const bf explicit_coeffs[MAX_EXPLICIT_COEFFS];
  const uint16_t col_idxs[MAX_FLAT_COL_IDXS];
  // I could bit-pack these but it's more trouble than it's worth
  const uchar2 num_linear_and_quadratic_terms_per_constraint[MAX_NON_BOOLEAN_CONSTRAINTS];
  // TODO: consider making this array for quadratic constraints only.
  // In practice there are relatively few linear constraints so it doesn't make much difference.
  const e2 decompression_factor;
  const e2 decompression_factor_squared;
  const e2 every_row_zerofier;
  const e2 omega_inv;
  const unsigned current_flat_col_idx;
  const unsigned current_flat_term_idx;
  const unsigned num_boolean_constraints;
  const unsigned num_non_boolean_quadratic_constraints;
  const unsigned num_non_boolean_constraints;
};

template <typename T>
DEVICE_FORCEINLINE void maybe_apply_coeff(const T &metadata, const unsigned coeff_idx, unsigned &explicit_coeff_idx, field::base_field &val) {
  switch (metadata.coeffs_info[coeff_idx]) {
  case COEFF_IS_ONE:
    break;
  case COEFF_IS_MINUS_ONE:
    val = field::base_field::neg(val);
    break;
  default:
    val = field::base_field::mul(val, metadata.explicit_coeffs[explicit_coeff_idx++]);
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void generic_constraints_kernel(__grid_constant__ const FlattenedGenericConstraintsMetadata metadata, matrix_getter<bf, ld_modifier::cg> witness_cols,
                                    matrix_getter<bf, ld_modifier::cg> memory_cols, vector_getter<e4, ld_modifier::ca> alphas,
                                    vectorized_e4_matrix_setter<st_modifier::cs> quotient, const unsigned log_n) {
  const unsigned n = 1 << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n)
    return;

  witness_cols.add_row(gid);
  memory_cols.add_row(gid);
  quotient.add_row(gid);

  e4 acc_linear{e4::zero()};
  e4 acc_quadratic{e4::zero()};

  // Boolean constraints
  for (unsigned constraint = 0; constraint < metadata.num_boolean_constraints; constraint++) {
    // generic boolean constraints should act on witness cols only (we assert this on the Rust side)
    const bf val_neg = bf::neg(witness_cols.get_at_col(metadata.col_idxs[constraint]));
    const bf val_squared = bf::mul(val_neg, val_neg);
    const e4 alpha_power = (alphas++).get();
    acc_quadratic = e4::add(acc_quadratic, e4::mul(alpha_power, val_squared));
    acc_linear = e4::add(acc_linear, e4::mul(alpha_power, val_neg));
  }

  unsigned flat_term_idx = 0;
  unsigned flat_col_idx = metadata.num_boolean_constraints;
  unsigned explicit_coeff_idx = 0;

  // Non-boolean quadratic constraints
  // Each contains at least one quadratic term and zero or more linear terms.
  for (unsigned constraint = 0; constraint < metadata.num_non_boolean_quadratic_constraints; constraint++) {
    const uchar2 num_linear_and_quadratic_terms = metadata.num_linear_and_quadratic_terms_per_constraint[constraint];
    const unsigned num_quadratic_terms = num_linear_and_quadratic_terms.x;
    const unsigned num_linear_terms = num_linear_and_quadratic_terms.y;

    bf quadratic_contribution{bf::zero()};
    unsigned lim = flat_term_idx + num_quadratic_terms;
    for (; flat_term_idx < lim; flat_term_idx++) {
      // Strangely, selecting between witness or memory cols incurs a 10-15% performance hit for this kernel
      // for n=2^22, but not for 2^21.
      // TODO: Double check performance for eventual production sizes.
      const bf val0 = get_witness_or_memory(metadata.col_idxs[flat_col_idx++], witness_cols, memory_cols);
      const bf val1 = get_witness_or_memory(metadata.col_idxs[flat_col_idx++], witness_cols, memory_cols);
      bf val = bf::mul(val0, val1);
      maybe_apply_coeff(metadata, flat_term_idx, explicit_coeff_idx, val);
      quadratic_contribution = bf::add(quadratic_contribution, val);
    }
    const e4 alpha_power = (alphas++).get();
    acc_quadratic = e4::add(acc_quadratic, e4::mul(alpha_power, quadratic_contribution));

    if (num_linear_terms > 0) {
      bf linear_contribution{bf::zero()};
      lim = flat_term_idx + num_linear_terms;
      for (; flat_term_idx < lim; flat_term_idx++) {
        bf val = get_witness_or_memory(metadata.col_idxs[flat_col_idx++], witness_cols, memory_cols);
        maybe_apply_coeff(metadata, flat_term_idx, explicit_coeff_idx, val);
        linear_contribution = bf::add(linear_contribution, val);
      }
      acc_linear = e4::add(acc_linear, e4::mul(alpha_power, linear_contribution));
    }
  }

  // Linear constraints
  for (unsigned constraint = metadata.num_non_boolean_quadratic_constraints; constraint < metadata.num_non_boolean_constraints; constraint++) {
    const uchar2 num_linear_and_quadratic_terms = metadata.num_linear_and_quadratic_terms_per_constraint[constraint];
    const unsigned num_linear_terms = num_linear_and_quadratic_terms.y;

    bf linear_contribution{bf::zero()};
    const unsigned lim = flat_term_idx + num_linear_terms;
    for (; flat_term_idx < lim; flat_term_idx++) {
      bf val = get_witness_or_memory(metadata.col_idxs[flat_col_idx++], witness_cols, memory_cols);
      maybe_apply_coeff(metadata, flat_term_idx, explicit_coeff_idx, val);
      linear_contribution = bf::add(linear_contribution, val);
    }

    const e4 alpha_power = (alphas++).get();
    acc_linear = e4::add(acc_linear, e4::mul(alpha_power, linear_contribution));
  }

  acc_quadratic = e4::mul(acc_quadratic, metadata.decompression_factor_squared);
  acc_linear = e4::mul(acc_linear, metadata.decompression_factor);
  e4 acc = e4::add(acc_quadratic, acc_linear);
  quotient.set(acc);
}

constexpr unsigned LOOKUP_VAL_IS_COL_FLAG = 255;

constexpr unsigned DELEGATED_MAX_WIDTH_3_LOOKUPS = 224;
constexpr unsigned DELEGATED_MAX_WIDTH_3_LOOKUP_VALS = 640;
constexpr unsigned DELEGATED_MAX_WIDTH_3_LOOKUP_COEFFS = 1408;
constexpr unsigned DELEGATED_MAX_WIDTH_3_LOOKUP_COLS = 1888;

extern "C" struct DelegatedWidth3LookupsLayout {
  const unsigned coeffs[DELEGATED_MAX_WIDTH_3_LOOKUP_COEFFS];
  const uint16_t col_idxs[DELEGATED_MAX_WIDTH_3_LOOKUP_COLS];
  const uint8_t num_terms_per_expression[DELEGATED_MAX_WIDTH_3_LOOKUP_VALS];
  const bool table_id_is_col[DELEGATED_MAX_WIDTH_3_LOOKUPS];
  const uint16_t e4_arg_cols[DELEGATED_MAX_WIDTH_3_LOOKUPS];
  const unsigned helpers_offset;
  const unsigned num_helpers_used;
  const unsigned num_lookups;
  const unsigned e4_arg_cols_start;
};

constexpr unsigned NON_DELEGATED_MAX_WIDTH_3_LOOKUPS = 24;
constexpr unsigned NON_DELEGATED_MAX_WIDTH_3_LOOKUP_VALS = 72;
constexpr unsigned NON_DELEGATED_MAX_WIDTH_3_LOOKUP_COEFFS = 32;
constexpr unsigned NON_DELEGATED_MAX_WIDTH_3_LOOKUP_COLS = 96;

extern "C" struct NonDelegatedWidth3LookupsLayout {
  const unsigned coeffs[NON_DELEGATED_MAX_WIDTH_3_LOOKUP_COEFFS];
  const uint16_t col_idxs[NON_DELEGATED_MAX_WIDTH_3_LOOKUP_COLS];
  const uint8_t num_terms_per_expression[NON_DELEGATED_MAX_WIDTH_3_LOOKUP_VALS];
  const bool table_id_is_col[NON_DELEGATED_MAX_WIDTH_3_LOOKUPS];
  const uint16_t e4_arg_cols[NON_DELEGATED_MAX_WIDTH_3_LOOKUPS];
  const unsigned helpers_offset;
  const unsigned num_helpers_used;
  const unsigned num_lookups;
  const unsigned e4_arg_cols_start;
};

template <typename T>
DEVICE_FORCEINLINE void enforce_width_3_lookup_args_construction(const T &layout, const matrix_getter<bf, ld_modifier::cg> &witness_cols,
                                                                 const matrix_getter<bf, ld_modifier::cg> &memory_cols,
                                                                 const vectorized_e4_matrix_getter<ld_modifier::cg> &stage_2_e4_cols,
                                                                 vector_getter<e4, ld_modifier::ca> &helpers, e4 &acc_quadratic) {
  unsigned col_idx = 0;
  unsigned val_idx = 0;
  unsigned coeff_idx = 0;
  for (unsigned term_idx = 0; term_idx < layout.num_lookups; term_idx++) {
    e4 acc = (helpers++).get();
    if (layout.table_id_is_col[term_idx]) {
      // Should be witness cols (we assert this on the Rust side)
      const bf id = witness_cols.get_at_col(layout.col_idxs[col_idx++]);
      acc = e4::add(acc, e4::mul((helpers++).get(), id));
    }
#pragma unroll
    for (unsigned j = 0; j < NUM_LOOKUP_ARGUMENT_KEY_PARTS - 1; j++) {
      const unsigned num_expr_terms = layout.num_terms_per_expression[val_idx++];
      if (num_expr_terms == LOOKUP_VAL_IS_COL_FLAG) {
        const bf val = get_witness_or_memory(layout.col_idxs[col_idx++], witness_cols, memory_cols);
        acc = e4::add(acc, e4::mul((helpers++).get(), val));
      } else {
        bf val{bf::zero()};
        const unsigned lim = col_idx + num_expr_terms;
        for (; col_idx < lim; col_idx++) {
          bf next = get_witness_or_memory(layout.col_idxs[col_idx], witness_cols, memory_cols);
          apply_coeff(layout.coeffs[coeff_idx++], next);
          val = bf::add(val, next);
        }
        if (num_expr_terms > 0) {
          acc = e4::add(acc, e4::mul((helpers++).get(), val));
        }
      }
    }
    const e4 e4_arg = stage_2_e4_cols.get_at_col(layout.e4_arg_cols[term_idx]);
    acc = e4::mul(acc, e4_arg);
    acc_quadratic = e4::add(acc_quadratic, acc);
  }
}

EXTERN __launch_bounds__(128, 8) __global__
    void delegated_width_3_lookups_kernel(__grid_constant__ const DelegatedWidth3LookupsLayout layout, matrix_getter<bf, ld_modifier::cg> witness_cols,
                                          matrix_getter<bf, ld_modifier::cg> memory_cols, vectorized_e4_matrix_getter<ld_modifier::cg> stage_2_e4_cols,
                                          vector_getter<e4, ld_modifier::ca> helpers,
                                          vectorized_e4_matrix_getter_setter<ld_modifier::cs, st_modifier::cs> quotient, const e2 decompression_factor_squared,
                                          const unsigned log_n) {
  const unsigned n = 1 << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n)
    return;

  witness_cols.add_row(gid);
  memory_cols.add_row(gid);
  stage_2_e4_cols.add_row(gid);
  quotient.add_row(gid);
  helpers += layout.helpers_offset;

  e4 acc_quadratic{e4::zero()};

  enforce_width_3_lookup_args_construction(layout, witness_cols, memory_cols, stage_2_e4_cols, helpers, acc_quadratic);

  acc_quadratic = e4::mul(acc_quadratic, decompression_factor_squared);
  const e4 current_quotient = quotient.get();
  acc_quadratic = e4::add(acc_quadratic, current_quotient);
  quotient.set(acc_quadratic);
}

// Assumes pred is a boolean (0 or 1) and enforces (pred - 1) * val == 0.
DEVICE_FORCEINLINE void enforce_val_zero_if_pred_zero(const bf predicate, const bf val, vector_getter<e4, ld_modifier::ca> &alphas, e4 &acc_quadratic,
                                                      e4 &acc_linear) {
  const e4 alpha_power = (alphas++).get();
  const bf prod = bf::mul(predicate, val);
  acc_quadratic = e4::add(acc_quadratic, e4::mul(alpha_power, prod));
  acc_linear = e4::add(acc_linear, e4::mul(alpha_power, bf::neg(val)));
}

DEVICE_FORCEINLINE void enforce_width_1_bf_arg_construction(const bf a, const bf b, const bf bf_arg, vector_getter<e4, ld_modifier::ca> &alphas,
                                                            vector_getter<e4, ld_modifier::ca> &helpers, e4 &acc_linear, e4 &acc_quadratic) {
  const e4 alpha = (alphas++).get();
  const bf prod = bf::mul(a, b);
  acc_quadratic = e4::add(acc_quadratic, e4::mul(alpha, prod));
  acc_linear = e4::add(acc_linear, e4::mul(alpha, bf::neg(bf_arg)));
}

DEVICE_FORCEINLINE void enforce_width_1_e4_arg_construction(const bf a, const bf b, const bf bf_arg, const unsigned e4_arg_idx,
                                                            const vectorized_e4_matrix_getter<ld_modifier::cg> &stage_2_e4_cols,
                                                            vector_getter<e4, ld_modifier::ca> &alphas, vector_getter<e4, ld_modifier::ca> &helpers,
                                                            e4 &acc_linear, e4 &acc_quadratic) {
  const e4 alpha = (alphas++).get();
  const bf sum = bf::add(a, b);
  // Thanks to precomputed helper factors, we get away with just one e4 x e4 mul.
  acc_linear = e4::add(acc_linear, e4::mul(alpha, bf::neg(sum)));
  const e4 alpha_times_gamma = (helpers++).get();
  const e4 alpha_times_gamma_squared_adjusted = (helpers++).get();
  const e4 bf_arg_term = e4::mul(alpha, bf_arg);
  const e4 gamma_terms = e4::add(alpha_times_gamma_squared_adjusted, e4::mul(alpha_times_gamma, sum));
  const e4 denoms_prod = e4::add(bf_arg_term, gamma_terms);
  const e4 e4_arg = stage_2_e4_cols.get_at_col(e4_arg_idx);
  const e4 quadratic_term = e4::mul(e4_arg, denoms_prod);
  acc_quadratic = e4::add(acc_quadratic, quadratic_term);
}

template <typename T>
DEVICE_FORCEINLINE void enforce_range_check_expressions_with_constant_terms(const T &expressions, unsigned &i, unsigned &expression_idx,
                                                                            unsigned &flat_term_idx, const matrix_getter<bf, ld_modifier::cg> &witness_cols,
                                                                            const matrix_getter<bf, ld_modifier::cg> &memory_cols,
                                                                            const matrix_getter<bf, ld_modifier::cg> &stage_2_bf_cols,
                                                                            const vectorized_e4_matrix_getter<ld_modifier::cg> &stage_2_e4_cols,
                                                                            const unsigned expression_pair_bound, vector_getter<e4, ld_modifier::ca> &alphas,
                                                                            vector_getter<e4, ld_modifier::ca> &helpers, e4 &acc_linear, e4 &acc_quadratic) {
#pragma unroll
  for (; i < expression_pair_bound; i++) {
    bf a_and_b[2];
    eval_a_and_b<false>(a_and_b, expressions, expression_idx, flat_term_idx, witness_cols, memory_cols, false);
    const bf a = a_and_b[0]; // not including constant contribution
    const bf b = a_and_b[1]; // not including constant contribution
    const bf bf_arg = stage_2_bf_cols.get_at_col(expressions.bf_dst_cols[i]);
    const e4 alpha = (alphas++).get();
    const bf prod = bf::mul(a, b);
    acc_quadratic = e4::add(acc_quadratic, e4::mul(alpha, prod));
    const bf a_constant_term = expressions.constant_terms[expression_idx - 2];
    const bf b_constant_term = expressions.constant_terms[expression_idx - 1];
    const bf linear_contribution_from_a_b_constants = bf::add(bf::mul(a, b_constant_term), bf::mul(b, a_constant_term));
    acc_linear = e4::add(acc_linear, e4::mul(alpha, bf::sub(linear_contribution_from_a_b_constants, bf_arg)));
    enforce_width_1_e4_arg_construction(a, b, bf_arg, expressions.e4_dst_cols[i], stage_2_e4_cols, alphas, helpers, acc_linear, acc_quadratic);
  }
}

extern "C" struct MultiplicitiesLayout {
  const unsigned src_cols_start;
  const unsigned dst_cols_start;
  const unsigned setup_cols_start;
  const unsigned num_dst_cols;
};

template <unsigned ENTRY_WIDTH>
DEVICE_FORCEINLINE void
enforce_lookup_multiplicities(const MultiplicitiesLayout &layout, const matrix_getter<bf, ld_modifier::cg> &setup_cols,
                              const matrix_getter<bf, ld_modifier::cg> &witness_cols, const vectorized_e4_matrix_getter<ld_modifier::cg> &stage_2_e4_cols,
                              vector_getter<e4, ld_modifier::ca> &alphas, vector_getter<e4, ld_modifier::ca> &helpers, e4 &acc_linear, e4 &acc_quadratic) {
  for (unsigned i = 0; i < layout.num_dst_cols; i++) {
    const e4 alpha = (alphas++).get();
    const bf m = witness_cols.get_at_col(layout.src_cols_start + i);
    acc_linear = e4::add(acc_linear, e4::mul(alpha, bf::neg(m)));
    e4 denom = (helpers++).get();
    const unsigned setup_cols_start = layout.setup_cols_start + i * ENTRY_WIDTH;
    denom = e4::add(denom, e4::mul(alpha, setup_cols.get_at_col(setup_cols_start)));
    if (ENTRY_WIDTH > 1) { // hint to compiler to optimize this out if possible
#pragma unroll
      for (unsigned i = 1; i < ENTRY_WIDTH; i++) {
        const e4 adjusted_linearization_challenge = (helpers++).get();
        const bf val = setup_cols.get_at_col(setup_cols_start + i);
        denom = e4::add(denom, e4::mul(adjusted_linearization_challenge, val));
      }
    }
    const e4 e4_arg = stage_2_e4_cols.get_at_col(layout.dst_cols_start + i);
    denom = e4::mul(denom, e4_arg);
    acc_quadratic = e4::add(acc_quadratic, denom);
  }
}

constexpr unsigned MAX_STATE_LINKAGE_CONSTRAINTS = 2;

extern "C" struct StateLinkageConstraints {
  const unsigned srcs[MAX_STATE_LINKAGE_CONSTRAINTS];
  const unsigned dsts[MAX_STATE_LINKAGE_CONSTRAINTS];
  const unsigned num_constraints;
};

constexpr bf SHIFT_16 = bf{1 << 16};

constexpr unsigned MAX_BOUNDARY_CONSTRAINTS_FIRST_ROW = 8;
constexpr unsigned MAX_BOUNDARY_CONSTRAINTS_ONE_BEFORE_LAST_ROW = 8;

extern "C" struct BoundaryConstraints {
  const unsigned first_row_cols[MAX_BOUNDARY_CONSTRAINTS_FIRST_ROW];
  const unsigned one_before_last_row_cols[MAX_BOUNDARY_CONSTRAINTS_ONE_BEFORE_LAST_ROW];
  const unsigned num_first_row;
  const unsigned num_one_before_last_row;
};

extern "C" struct ConstantsTimesChallenges {
  const e4 first_row;
  const e4 one_before_last_row;
  const e4 sum;
};

// TODO once constraints are done
//  - think about the most sensible way to split them up into multiple kernels.
//    e.g. one kernel for memory-col-heavy terms and one kernel for witness-col-heavy terms.
//  - Turn e4::sub contributions to acc_linear into e4::adds and negate acc_linear once at the end
EXTERN __launch_bounds__(128, 8) __global__ void hardcoded_constraints_kernel(
    matrix_getter<bf, ld_modifier::cg> setup_cols, matrix_getter<bf, ld_modifier::cg> witness_cols, matrix_getter<bf, ld_modifier::cg> memory_cols,
    matrix_getter<bf, ld_modifier::cg> stage_2_bf_cols, vectorized_e4_matrix_getter<ld_modifier::cg> stage_2_e4_cols, const bool process_delegations,
    const bool handle_delegation_requests, const unsigned delegation_aux_poly_col, __grid_constant__ const DelegationChallenges delegation_challenges,
    __grid_constant__ const DelegationProcessingMetadata delegation_processing_metadata,
    __grid_constant__ const DelegationRequestMetadata delegation_request_metadata, const unsigned memory_args_start, const unsigned memory_grand_product_col,
    __grid_constant__ const LazyInitTeardownLayout lazy_init_teardown_layout, __grid_constant__ const ShuffleRamAccesses shuffle_ram_accesses,
    const bool process_batch_ram_access, __grid_constant__ const BatchedRamAccesses batched_ram_accesses, const bool process_registers_and_indirect_access,
    __grid_constant__ const RegisterAndIndirectAccesses register_and_indirect_accesses, __grid_constant__ const RangeCheckArgsLayout range_check_16_layout,
    __grid_constant__ const FlattenedLookupExpressionsLayout expressions,
    __grid_constant__ const FlattenedLookupExpressionsForShuffleRamLayout expressions_for_shuffle_ram,
    __grid_constant__ const NonDelegatedWidth3LookupsLayout width_3_lookups_layout,
    __grid_constant__ const MultiplicitiesLayout range_check_16_multiplicities_layout,
    __grid_constant__ const MultiplicitiesLayout timestamp_range_check_multiplicities_layout,
    __grid_constant__ const MultiplicitiesLayout generic_lookup_multiplicities_layout,
    __grid_constant__ const StateLinkageConstraints state_linkage_constraints, __grid_constant__ const BoundaryConstraints boundary_constraints,
    vector_getter<e4, ld_modifier::ca> alphas, vector_getter<e4, ld_modifier::ca> alphas_every_row_except_last_two, vector_getter<e4, ld_modifier::ca> betas,
    vector_getter<e4, ld_modifier::ca> helpers, const ConstantsTimesChallenges *constants_times_challenges,
    vectorized_e4_matrix_getter_setter<ld_modifier::cs, st_modifier::cs> quotient, const bf memory_timestamp_high_from_circuit_idx,
    const e2 decompression_factor, const e2 decompression_factor_squared, const e2 every_row_zerofier, const e2 omega_inv, const e2 omega_inv_squared,
    const unsigned log_n) {
  const unsigned n = 1 << log_n;
  const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n)
    return;

  setup_cols.add_row(gid);
  witness_cols.add_row(gid);
  memory_cols.add_row(gid);
  stage_2_bf_cols.add_row(gid);
  stage_2_e4_cols.add_row(gid);
  quotient.add_row(gid);

  e4 acc_linear{e4::zero()};
  e4 acc_quadratic{e4::zero()};

  // TODO: consider factoring out the predicate from some of the sequences below, accumulating to a temporary acc_quadratic,
  // and multiplying the temporary acc_quadratic by predicate at the end of each sequence.
  if (process_delegations) {
    const auto &metadata = delegation_processing_metadata;
    const bf predicate = memory_cols.get_at_col(metadata.multiplicity_col);
    const bf vals[4] = {predicate, memory_cols.get_at_col(metadata.abi_mem_offset_high_col), memory_cols.get_at_col(metadata.write_timestamp_col),
                        memory_cols.get_at_col(metadata.write_timestamp_col + 1)};
    // the first iteration enforces that predicate is a boolean. conveniently, this can use the same function.
#pragma unroll
    for (unsigned i = 0; i < 4; i++)
      enforce_val_zero_if_pred_zero(predicate, vals[i], alphas, acc_quadratic, acc_linear);

    if (process_batch_ram_access) {
#pragma unroll
      for (unsigned i = 0; i < batched_ram_accesses.num_accesses; i++) {
        const auto &access = batched_ram_accesses.accesses[i];
        enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_timestamp_col), alphas, acc_quadratic, acc_linear);
        enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_timestamp_col + 1), alphas, acc_quadratic, acc_linear);
        enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_value_col), alphas, acc_quadratic, acc_linear);
        enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_value_col + 1), alphas, acc_quadratic, acc_linear);
        if (access.is_write) {
          enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.maybe_write_value_col), alphas, acc_quadratic, acc_linear);
          enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.maybe_write_value_col + 1), alphas, acc_quadratic, acc_linear);
        }
      }
    }

    if (process_registers_and_indirect_access) {
      unsigned flat_indirect_idx = 0;
#pragma unroll
      for (unsigned i = 0; i < register_and_indirect_accesses.num_register_accesses; i++) {
        {
          const auto &access = register_and_indirect_accesses.register_accesses[i];
          enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_timestamp_col), alphas, acc_quadratic, acc_linear);
          enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_timestamp_col + 1), alphas, acc_quadratic, acc_linear);
          enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_value_col), alphas, acc_quadratic, acc_linear);
          enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_value_col + 1), alphas, acc_quadratic, acc_linear);
          if (access.is_write) {
            enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.maybe_write_value_col), alphas, acc_quadratic, acc_linear);
            enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.maybe_write_value_col + 1), alphas, acc_quadratic, acc_linear);
          }
        }
        const unsigned num_indirect_accesses = register_and_indirect_accesses.indirect_accesses_per_register_access[i];
#pragma unroll 1
        for (unsigned j = 0; j < num_indirect_accesses; j++, flat_indirect_idx++) {
          const auto &access = register_and_indirect_accesses.indirect_accesses[flat_indirect_idx];
          enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_timestamp_col), alphas, acc_quadratic, acc_linear);
          enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_timestamp_col + 1), alphas, acc_quadratic, acc_linear);
          enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_value_col), alphas, acc_quadratic, acc_linear);
          enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.read_value_col + 1), alphas, acc_quadratic, acc_linear);
          if (access.is_write) {
            enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.maybe_write_value_col), alphas, acc_quadratic, acc_linear);
            enforce_val_zero_if_pred_zero(predicate, memory_cols.get_at_col(access.maybe_write_value_col + 1), alphas, acc_quadratic, acc_linear);
          }
          if (j > 0 && access.address_derivation_carry_bit_num_elements > 0) {
            // Boolean check for carry bit
            const bf carry_bit = memory_cols.get_at_col(access.address_derivation_carry_bit_col);
            enforce_val_zero_if_pred_zero(carry_bit, carry_bit, alphas, acc_quadratic, acc_linear);
          }
        }
      }
    }
  }

  // Range check 16 and timestamp range check args
  {
#pragma unroll
    for (unsigned i = 0; i < range_check_16_layout.num_dst_cols; i++) {
      const unsigned src = 2 * i + range_check_16_layout.src_cols_start;
      const bf a = witness_cols.get_at_col(src);
      const bf b = witness_cols.get_at_col(src + 1);
      const bf bf_arg = stage_2_bf_cols.get_at_col(range_check_16_layout.bf_args_start + i);
      enforce_width_1_bf_arg_construction(a, b, bf_arg, alphas, helpers, acc_linear, acc_quadratic);
      enforce_width_1_e4_arg_construction(a, b, bf_arg, range_check_16_layout.e4_args_start + i, stage_2_e4_cols, alphas, helpers, acc_linear, acc_quadratic);
    }

    unsigned i{0}, expression_idx{0}, flat_term_idx{0};

    if (expressions.range_check_16_constant_terms_are_zero) {
#pragma unroll
      for (; i < expressions.num_range_check_16_expression_pairs; i++) {
        bf a_and_b[2];
        eval_a_and_b<false>(a_and_b, expressions, expression_idx, flat_term_idx, witness_cols, memory_cols, true);
        const bf bf_arg = stage_2_bf_cols.get_at_col(expressions.bf_dst_cols[i]);
        enforce_width_1_bf_arg_construction(a_and_b[0], a_and_b[1], bf_arg, alphas, helpers, acc_linear, acc_quadratic);
        enforce_width_1_e4_arg_construction(a_and_b[0], a_and_b[1], bf_arg, expressions.e4_dst_cols[i], stage_2_e4_cols, alphas, helpers, acc_linear,
                                            acc_quadratic);
      }
    } else {
      enforce_range_check_expressions_with_constant_terms(expressions, i, expression_idx, flat_term_idx, witness_cols, memory_cols, stage_2_bf_cols,
                                                          stage_2_e4_cols, expressions.num_range_check_16_expression_pairs, alphas, helpers, acc_linear,
                                                          acc_quadratic);
    }

    if (lazy_init_teardown_layout.process_shuffle_ram_init) {
      const bf a = memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start);
      const bf b = memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start + 1);
      const bf bf_arg = stage_2_bf_cols.get_at_col(lazy_init_teardown_layout.bf_arg_col);
      enforce_width_1_bf_arg_construction(a, b, bf_arg, alphas, helpers, acc_linear, acc_quadratic);
      enforce_width_1_e4_arg_construction(a, b, bf_arg, lazy_init_teardown_layout.e4_arg_col, stage_2_e4_cols, alphas, helpers, acc_linear, acc_quadratic);
    }

    if (expressions.timestamp_constant_terms_are_zero) {
      const unsigned expression_pair_bound = i + expressions.num_timestamp_expression_pairs;
#pragma unroll
      for (; i < expression_pair_bound; i++) {
        bf a_and_b[2];
        eval_a_and_b<false>(a_and_b, expressions, expression_idx, flat_term_idx, witness_cols, memory_cols, true);
        const bf bf_arg = stage_2_bf_cols.get_at_col(expressions.bf_dst_cols[i]);
        enforce_width_1_bf_arg_construction(a_and_b[0], a_and_b[1], bf_arg, alphas, helpers, acc_linear, acc_quadratic);
        enforce_width_1_e4_arg_construction(a_and_b[0], a_and_b[1], bf_arg, expressions.e4_dst_cols[i], stage_2_e4_cols, alphas, helpers, acc_linear,
                                            acc_quadratic);
      }
    } else {
      const unsigned expression_pair_bound = i + expressions.num_timestamp_expression_pairs;
      enforce_range_check_expressions_with_constant_terms(expressions, i, expression_idx, flat_term_idx, witness_cols, memory_cols, stage_2_bf_cols,
                                                          stage_2_e4_cols, expression_pair_bound, alphas, helpers, acc_linear, acc_quadratic);
    }

    // TODO (optional): If i add a spurious "setup_cols" argument to the eval_a_and_b overload for non-shuffle-ram expressions,
    // I could use enforce_range_check_expressions_with_constant_terms here too.
#pragma unroll
    for (unsigned i = 0, expression_idx = 0, flat_term_idx = 0; i < expressions_for_shuffle_ram.num_expression_pairs; i++) {
      bf a_and_b[2];
      eval_a_and_b<false>(a_and_b, expressions_for_shuffle_ram, expression_idx, flat_term_idx, setup_cols, witness_cols, memory_cols);
      const bf a = a_and_b[0]; // not including constant contribution
      const bf b = a_and_b[1]; // not including constant contribution
      const bf bf_arg = stage_2_bf_cols.get_at_col(expressions_for_shuffle_ram.bf_dst_cols[i]);
      const e4 alpha = (alphas++).get();
      const bf prod = bf::mul(a, b);
      acc_quadratic = e4::add(acc_quadratic, e4::mul(alpha, prod));
      const bf a_constant_term = expressions_for_shuffle_ram.constant_terms[expression_idx - 2];
      const bf b_constant_term = expressions_for_shuffle_ram.constant_terms[expression_idx - 1];
      const bf b_constant_term_adjusted = bf::sub(b_constant_term, memory_timestamp_high_from_circuit_idx);
      const bf linear_contribution_from_a_b_constants = bf::add(bf::mul(a, b_constant_term_adjusted), bf::mul(b, a_constant_term));
      acc_linear = e4::add(acc_linear, e4::mul(alpha, bf::sub(linear_contribution_from_a_b_constants, bf_arg)));
      enforce_width_1_e4_arg_construction(a, b, bf_arg, expressions_for_shuffle_ram.e4_dst_cols[i], stage_2_e4_cols, alphas, helpers, acc_linear,
                                          acc_quadratic);
    }
  }

  if (process_delegations) {
    // width 3 lookups were already handled by delegated_width_3_lookups_kernel.
    // width_3_lookups_layout is just a placeholder with enough info to account for the alphas and helpers the other kernel used.
    alphas += width_3_lookups_layout.num_lookups;
    helpers += width_3_lookups_layout.num_helpers_used;
  } else {
    enforce_width_3_lookup_args_construction(width_3_lookups_layout, witness_cols, memory_cols, stage_2_e4_cols, helpers, acc_quadratic);
    alphas += width_3_lookups_layout.num_lookups;
  }

  enforce_lookup_multiplicities<1>(range_check_16_multiplicities_layout, setup_cols, witness_cols, stage_2_e4_cols, alphas, helpers, acc_linear, acc_quadratic);
  enforce_lookup_multiplicities<1>(timestamp_range_check_multiplicities_layout, setup_cols, witness_cols, stage_2_e4_cols, alphas, helpers, acc_linear,
                                   acc_quadratic);
  enforce_lookup_multiplicities<NUM_LOOKUP_ARGUMENT_KEY_PARTS>(generic_lookup_multiplicities_layout, setup_cols, witness_cols, stage_2_e4_cols, alphas, helpers,
                                                               acc_linear, acc_quadratic);

  if (handle_delegation_requests) {
    const auto &metadata = delegation_request_metadata;
    const bf m = memory_cols.get_at_col(metadata.multiplicity_col);
    const e4 alpha = (alphas++).get();
    acc_linear = e4::add(acc_linear, e4::mul(alpha, bf::neg(m)));
    e4 denom = (helpers++).get();
    denom = e4::add(denom, e4::mul(alpha, memory_cols.get_at_col(metadata.delegation_type_col)));
    denom = e4::add(denom, e4::mul((helpers++).get(), memory_cols.get_at_col(metadata.abi_mem_offset_high_col)));
    denom = e4::add(denom, e4::mul((helpers++).get(), setup_cols.get_at_col(metadata.timestamp_setup_col)));
    denom = e4::add(denom, e4::mul((helpers++).get(), setup_cols.get_at_col(metadata.timestamp_setup_col + 1)));
    const e4 e4_arg = stage_2_e4_cols.get_at_col(delegation_aux_poly_col);
    acc_quadratic = e4::add(acc_quadratic, e4::mul(e4_arg, denom));
  }

  if (process_delegations) {
    const auto &metadata = delegation_processing_metadata;
    const bf m = memory_cols.get_at_col(metadata.multiplicity_col);
    const e4 alpha = (alphas++).get();
    acc_linear = e4::add(acc_linear, e4::mul(alpha, bf::neg(m)));
    e4 denom = (helpers++).get();
    denom = e4::add(denom, e4::mul((helpers++).get(), memory_cols.get_at_col(metadata.abi_mem_offset_high_col)));
    denom = e4::add(denom, e4::mul((helpers++).get(), memory_cols.get_at_col(metadata.write_timestamp_col)));
    denom = e4::add(denom, e4::mul((helpers++).get(), memory_cols.get_at_col(metadata.write_timestamp_col + 1)));
    const e4 e4_arg = stage_2_e4_cols.get_at_col(delegation_aux_poly_col);
    acc_quadratic = e4::add(acc_quadratic, e4::mul(e4_arg, denom));
  }

  if (lazy_init_teardown_layout.process_shuffle_ram_init) {

    e4 e4_arg_prev{};
    {
      const bf address_low = memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start);
      const bf address_high = memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start + 1);
      const bf value_low = memory_cols.get_at_col(lazy_init_teardown_layout.teardown_value_start);
      const bf value_high = memory_cols.get_at_col(lazy_init_teardown_layout.teardown_value_start + 1);
      const bf timestamp_low = memory_cols.get_at_col(lazy_init_teardown_layout.teardown_timestamp_start);
      const bf timestamp_high = memory_cols.get_at_col(lazy_init_teardown_layout.teardown_timestamp_start + 1);
      const bf final_borrow = witness_cols.get_at_col(lazy_init_teardown_layout.init_address_final_borrow);

      // First, enforce lazy init address, value, and timestamp limbs are zero if "final borrow" is zero
      enforce_val_zero_if_pred_zero(final_borrow, address_low, alphas, acc_quadratic, acc_linear);
      enforce_val_zero_if_pred_zero(final_borrow, address_high, alphas, acc_quadratic, acc_linear);
      enforce_val_zero_if_pred_zero(final_borrow, value_low, alphas, acc_quadratic, acc_linear);
      enforce_val_zero_if_pred_zero(final_borrow, value_high, alphas, acc_quadratic, acc_linear);
      enforce_val_zero_if_pred_zero(final_borrow, timestamp_low, alphas, acc_quadratic, acc_linear);
      enforce_val_zero_if_pred_zero(final_borrow, timestamp_high, alphas, acc_quadratic, acc_linear);

      // now, enforce the lazy init arg itself
      e4 numerator = e4::mul((helpers++).get(), address_low);
      numerator = e4::add(numerator, e4::mul((helpers++).get(), address_high));
      acc_linear = e4::sub(acc_linear, numerator);

      e4 denom{numerator};
      denom = e4::add(denom, e4::mul((helpers++).get(), value_low));
      denom = e4::add(denom, e4::mul((helpers++).get(), value_high));
      denom = e4::add(denom, e4::mul((helpers++).get(), timestamp_low));
      denom = e4::add(denom, e4::mul((helpers++).get(), timestamp_high));

      const e4 alpha_times_gamma_adjusted = (helpers++).get();
      denom = e4::add(denom, alpha_times_gamma_adjusted);

      const e4 e4_arg = stage_2_e4_cols.get_at_col(memory_args_start);
      acc_quadratic = e4::add(acc_quadratic, e4::mul(e4_arg, denom));
      e4_arg_prev = e4_arg;

      alphas++; // advance alpha explicitly
    }

    // Some write timestamp limb contributions are common across accesses:
    const bf write_timestamp_in_setup_low = setup_cols.get_at_col(shuffle_ram_accesses.write_timestamp_in_setup_start);
    const bf write_timestamp_in_setup_high = setup_cols.get_at_col(shuffle_ram_accesses.write_timestamp_in_setup_start + 1);
#pragma unroll 1
    for (unsigned i = 0; i < shuffle_ram_accesses.num_accesses; i++) {
      const auto &access = shuffle_ram_accesses.accesses[i];

      const bf address_low = memory_cols.get_at_col(access.address_start);
      e4 numerator = e4::mul((helpers++).get(), address_low);

      if (access.is_register_only) {
        alphas++; // constant bf::one() is already accounted for in numerator constant helper
      } else {
        const bf address_high = memory_cols.get_at_col(access.address_start + 1);
        numerator = e4::add(numerator, e4::mul((helpers++).get(), address_high));
        numerator = e4::add(numerator, e4::mul((alphas++).get(), memory_cols.get_at_col(access.maybe_is_register_start)));
      }

      e4 denom{};

      const e4 value_low_helper = (helpers++).get();
      const e4 value_high_helper = (helpers++).get();
      if (access.is_write) {
        denom = numerator;

        const bf read_value_low = memory_cols.get_at_col(access.read_value_start);
        denom = e4::add(denom, e4::mul(value_low_helper, read_value_low));
        const bf read_value_high = memory_cols.get_at_col(access.read_value_start + 1);
        denom = e4::add(denom, e4::mul(value_high_helper, read_value_high));

        const bf write_value_low = memory_cols.get_at_col(access.maybe_write_value_start);
        numerator = e4::add(numerator, e4::mul(value_low_helper, write_value_low));
        const bf write_value_high = memory_cols.get_at_col(access.maybe_write_value_start + 1);
        numerator = e4::add(numerator, e4::mul(value_high_helper, write_value_high));
      } else {
        const bf value_low = memory_cols.get_at_col(access.read_value_start);
        numerator = e4::add(numerator, e4::mul(value_low_helper, value_low));
        const bf value_high = memory_cols.get_at_col(access.read_value_start + 1);
        numerator = e4::add(numerator, e4::mul(value_high_helper, value_high));

        denom = numerator;
      }

      const e4 timestamp_low_helper = (helpers++).get();
      const e4 timestamp_high_helper = (helpers++).get();

      const bf read_timestamp_low = memory_cols.get_at_col(access.read_timestamp_start);
      denom = e4::add(denom, e4::mul(timestamp_low_helper, read_timestamp_low));
      const bf read_timestamp_high = memory_cols.get_at_col(access.read_timestamp_start + 1);
      denom = e4::add(denom, e4::mul(timestamp_high_helper, read_timestamp_high));

      numerator = e4::add(numerator, e4::mul(timestamp_low_helper, write_timestamp_in_setup_low));
      numerator = e4::add(numerator, e4::mul(timestamp_high_helper, write_timestamp_in_setup_high));

      // adjusted constant contributions
      denom = e4::add(denom, (helpers++).get());
      numerator = e4::add(numerator, (helpers++).get());

      const e4 e4_arg = stage_2_e4_cols.get_at_col(memory_args_start + 1 + i);
      acc_quadratic = e4::add(acc_quadratic, e4::mul(e4_arg, denom));

      acc_quadratic = e4::sub(acc_quadratic, e4::mul(e4_arg_prev, numerator));
      e4_arg_prev = e4_arg;
    }
  }

  if (process_batch_ram_access) {
    // address_high and write_timestamp limbs are which is common across accesses
    const bf address_high = memory_cols.get_at_col(batched_ram_accesses.abi_mem_offset_high_col);
    const bf write_timestamp_low = memory_cols.get_at_col(batched_ram_accesses.write_timestamp_col);
    const bf write_timestamp_high = memory_cols.get_at_col(batched_ram_accesses.write_timestamp_col + 1);
#pragma unroll 1
    for (unsigned i = 0; i < batched_ram_accesses.num_accesses; i++) {
      const auto &access = batched_ram_accesses.accesses[i];

      e4 numerator = e4::mul((helpers++).get(), address_high);

      e4 denom{};

      const e4 value_low_helper = (helpers++).get();
      const e4 value_high_helper = (helpers++).get();
      if (access.is_write) {
        denom = numerator;

        const bf read_value_low = memory_cols.get_at_col(access.read_value_col);
        denom = e4::add(denom, e4::mul(value_low_helper, read_value_low));
        const bf read_value_high = memory_cols.get_at_col(access.read_value_col + 1);
        denom = e4::add(denom, e4::mul(value_high_helper, read_value_high));

        const bf write_value_low = memory_cols.get_at_col(access.maybe_write_value_col);
        numerator = e4::add(numerator, e4::mul(value_low_helper, write_value_low));
        const bf write_value_high = memory_cols.get_at_col(access.maybe_write_value_col + 1);
        numerator = e4::add(numerator, e4::mul(value_high_helper, write_value_high));
      } else {
        const bf value_low = memory_cols.get_at_col(access.read_value_col);
        numerator = e4::add(numerator, e4::mul(value_low_helper, value_low));
        const bf value_high = memory_cols.get_at_col(access.read_value_col + 1);
        numerator = e4::add(numerator, e4::mul(value_high_helper, value_high));

        denom = numerator;
      }

      const e4 timestamp_low_helper = (helpers++).get();
      const e4 timestamp_high_helper = (helpers++).get();

      numerator = e4::add(numerator, e4::mul(timestamp_low_helper, write_timestamp_low));
      numerator = e4::add(numerator, e4::mul(timestamp_high_helper, write_timestamp_high));

      const bf read_timestamp_low = memory_cols.get_at_col(access.read_timestamp_col);
      denom = e4::add(denom, e4::mul(timestamp_low_helper, read_timestamp_low));
      const bf read_timestamp_high = memory_cols.get_at_col(access.read_timestamp_col + 1);
      denom = e4::add(denom, e4::mul(timestamp_high_helper, read_timestamp_high));

      // adjusted constant contributions
      const e4 constant = (helpers++).get();
      denom = e4::add(denom, constant);
      const e4 e4_arg = stage_2_e4_cols.get_at_col(memory_args_start + i);
      acc_quadratic = e4::add(acc_quadratic, e4::mul(e4_arg, denom));

      // flush result
      if (i == 0) {
        acc_linear = e4::sub(acc_linear, numerator);
      } else {
        numerator = e4::add(numerator, constant);

        // TODO: When you have a batch ram test case again, move e4_arg_prev to registers,
        // as it is for the other two access types.
        const e4 e4_arg_prev = stage_2_e4_cols.get_at_col(memory_args_start + i - 1);
        acc_quadratic = e4::sub(acc_quadratic, e4::mul(e4_arg_prev, numerator));
      }
    }

    alphas += batched_ram_accesses.num_accesses;
  }

  if (process_registers_and_indirect_access) {
    const bf write_timestamp_low = memory_cols.get_at_col(register_and_indirect_accesses.write_timestamp_col);
    const bf write_timestamp_high = memory_cols.get_at_col(register_and_indirect_accesses.write_timestamp_col + 1);
    unsigned flat_indirect_idx = 0;
    e4 e4_arg_prev{};
#pragma unroll 1
    for (unsigned i = 0; i < register_and_indirect_accesses.num_register_accesses; i++) {
      bf base_low;
      bf base_high;
      {
        const auto &access = register_and_indirect_accesses.register_accesses[i];
        e4 numerator{};
        e4 denom{};

        const e4 value_low_helper = (helpers++).get();
        const e4 value_high_helper = (helpers++).get();
        if (access.is_write) {
          const bf read_value_low = memory_cols.get_at_col(access.read_value_col);
          denom = e4::mul(value_low_helper, read_value_low);
          const bf read_value_high = memory_cols.get_at_col(access.read_value_col + 1);
          denom = e4::add(denom, e4::mul(value_high_helper, read_value_high));

          // imitate arg construction
          base_low = bf::into_canonical(read_value_low);
          base_high = bf::into_canonical(read_value_high);

          const bf write_value_low = memory_cols.get_at_col(access.maybe_write_value_col);
          numerator = e4::mul(value_low_helper, write_value_low);
          const bf write_value_high = memory_cols.get_at_col(access.maybe_write_value_col + 1);
          numerator = e4::add(numerator, e4::mul(value_high_helper, write_value_high));
        } else {
          const bf value_low = memory_cols.get_at_col(access.read_value_col);
          numerator = e4::mul(value_low_helper, value_low);
          const bf value_high = memory_cols.get_at_col(access.read_value_col + 1);
          numerator = e4::add(numerator, e4::mul(value_high_helper, value_high));

          // imitate arg construction
          base_low = bf::into_canonical(value_low);
          base_high = bf::into_canonical(value_high);

          denom = numerator;
        }

        const e4 timestamp_low_helper = (helpers++).get();
        const e4 timestamp_high_helper = (helpers++).get();

        numerator = e4::add(numerator, e4::mul(timestamp_low_helper, write_timestamp_low));
        numerator = e4::add(numerator, e4::mul(timestamp_high_helper, write_timestamp_high));

        const bf read_timestamp_low = memory_cols.get_at_col(access.read_timestamp_col);
        denom = e4::add(denom, e4::mul(timestamp_low_helper, read_timestamp_low));
        const bf read_timestamp_high = memory_cols.get_at_col(access.read_timestamp_col + 1);
        denom = e4::add(denom, e4::mul(timestamp_high_helper, read_timestamp_high));

        // adjusted constant contributions
        const e4 constant = (helpers++).get();
        denom = e4::add(denom, constant);
        const e4 e4_arg = stage_2_e4_cols.get_at_col(memory_args_start + i + flat_indirect_idx);
        acc_quadratic = e4::add(acc_quadratic, e4::mul(e4_arg, denom));

        // flush result
        if (i == 0) {
          acc_linear = e4::sub(acc_linear, numerator);
          e4_arg_prev = e4_arg;
        } else {
          numerator = e4::add(numerator, constant);
          acc_quadratic = e4::sub(acc_quadratic, e4::mul(e4_arg_prev, numerator));
          e4_arg_prev = e4_arg;
        }
      }

      const unsigned start = flat_indirect_idx;
      const unsigned end = flat_indirect_idx + register_and_indirect_accesses.indirect_accesses_per_register_access[i];
#pragma unroll 1
      for (; flat_indirect_idx < end; flat_indirect_idx++) {
        const auto &access = register_and_indirect_accesses.indirect_accesses[flat_indirect_idx];
        e4 numerator{};
        e4 denom{};

        const e4 address_low_helper = (helpers++).get();
        const e4 address_high_helper = (helpers++).get();
        if (flat_indirect_idx == start || access.address_derivation_carry_bit_num_elements == 0) {
          numerator = e4::mul(address_low_helper, base_low);
          numerator = e4::add(numerator, e4::mul(address_high_helper, base_high));
        } else {
          const bf carry_bit = memory_cols.get_at_col(access.address_derivation_carry_bit_col);
          numerator = e4::mul(address_low_helper, bf::sub(base_low, bf::mul(carry_bit, SHIFT_16)));
          numerator = e4::add(numerator, e4::mul(address_high_helper, bf::add(base_high, carry_bit)));
        }

        const e4 value_low_helper = (helpers++).get();
        const e4 value_high_helper = (helpers++).get();
        if (access.is_write) {
          denom = numerator;

          const bf read_value_low = memory_cols.get_at_col(access.read_value_col);
          denom = e4::add(denom, e4::mul(value_low_helper, read_value_low));
          const bf read_value_high = memory_cols.get_at_col(access.read_value_col + 1);
          denom = e4::add(denom, e4::mul(value_high_helper, read_value_high));

          const bf write_value_low = memory_cols.get_at_col(access.maybe_write_value_col);
          numerator = e4::add(numerator, e4::mul(value_low_helper, write_value_low));
          const bf write_value_high = memory_cols.get_at_col(access.maybe_write_value_col + 1);
          numerator = e4::add(numerator, e4::mul(value_high_helper, write_value_high));
        } else {
          const bf value_low = memory_cols.get_at_col(access.read_value_col);
          numerator = e4::add(numerator, e4::mul(value_low_helper, value_low));
          const bf value_high = memory_cols.get_at_col(access.read_value_col + 1);
          numerator = e4::add(numerator, e4::mul(value_high_helper, value_high));

          denom = numerator;
        }

        const e4 timestamp_low_helper = (helpers++).get();
        const e4 timestamp_high_helper = (helpers++).get();

        numerator = e4::add(numerator, e4::mul(timestamp_low_helper, write_timestamp_low));
        numerator = e4::add(numerator, e4::mul(timestamp_high_helper, write_timestamp_high));

        const bf read_timestamp_low = memory_cols.get_at_col(access.read_timestamp_col);
        denom = e4::add(denom, e4::mul(timestamp_low_helper, read_timestamp_low));
        const bf read_timestamp_high = memory_cols.get_at_col(access.read_timestamp_col + 1);
        denom = e4::add(denom, e4::mul(timestamp_high_helper, read_timestamp_high));

        // adjusted constant contributions
        const e4 constant = (helpers++).get();
        denom = e4::add(denom, constant);
        const e4 e4_arg = stage_2_e4_cols.get_at_col(memory_args_start + flat_indirect_idx + i + 1);
        acc_quadratic = e4::add(acc_quadratic, e4::mul(e4_arg, denom));

        // flush result
        numerator = e4::add(numerator, constant);
        acc_quadratic = e4::sub(acc_quadratic, e4::mul(e4_arg_prev, numerator));
        e4_arg_prev = e4_arg;
      }
    }

    alphas += register_and_indirect_accesses.num_register_accesses + flat_indirect_idx;
  }

  {
    // kinda ugly with 3 e4 x e4 muls, but hopefully negligible overall
    const e4 memory_arg_entry = stage_2_e4_cols.get_at_col(memory_grand_product_col - 1);
    const e4 grand_product_entry = stage_2_e4_cols.get_at_col(memory_grand_product_col);
    e4 grand_product_entry_next{};
    if (gid == n - 1) {
      stage_2_e4_cols.sub_row(gid);
      grand_product_entry_next = stage_2_e4_cols.get_at_col(memory_grand_product_col);
      stage_2_e4_cols.add_row(gid);
    } else {
      stage_2_e4_cols.add_row(1);
      grand_product_entry_next = stage_2_e4_cols.get_at_col(memory_grand_product_col);
      stage_2_e4_cols.sub_row(1);
    }
    const e4 alpha = (alphas++).get();
    acc_linear = e4::add(acc_linear, e4::mul(alpha, grand_product_entry_next));
    const e4 prod = e4::mul(memory_arg_entry, grand_product_entry);
    acc_quadratic = e4::sub(acc_quadratic, e4::mul(alpha, prod));
  }

  // Finalize "every row except last" contributions
  acc_quadratic = e4::mul(acc_quadratic, decompression_factor_squared);
  acc_linear = e4::mul(acc_linear, decompression_factor);
  e4 acc = e4::add(acc_quadratic, acc_linear);
  const e4 current_quotient = quotient.get();
  acc = e4::add(acc, current_quotient);
  acc = e4::add(acc, constants_times_challenges->sum);
  const unsigned shift = 1 << (field::CIRCLE_GROUP_LOG_ORDER - log_n - 1);
  const e2 x = get_power_of_w(shift * (2 * gid + 1), false);
  const e2 num = e2::sub(x, omega_inv);
  e2 multiplier = e2::mul(num, every_row_zerofier);
  acc = e4::mul(acc, multiplier);
  // TODO: fold beta powers into corresponding alpha powers
  acc = e4::mul(acc, betas.get(5));

  // Constraints at every row except last two
  if (state_linkage_constraints.num_constraints > 0 || lazy_init_teardown_layout.process_shuffle_ram_init) {
    e4 acc_linear{e4::zero()};

    {
      auto witness_cols_next_row = witness_cols.copy();
      if (gid < n - 1)
        witness_cols_next_row.add_row(1);
      else
        witness_cols_next_row.sub_row(gid);

      for (unsigned i = 0; i < state_linkage_constraints.num_constraints; i++) {
        const e4 alpha = (alphas_every_row_except_last_two++).get();
        const bf src_val = witness_cols.get_at_col(state_linkage_constraints.srcs[i]);
        const bf dst_val = witness_cols_next_row.get_at_col(state_linkage_constraints.dsts[i]);
        acc_linear = e4::add(acc_linear, e4::mul(alpha, bf::sub(src_val, dst_val)));
      }
    }

    if (lazy_init_teardown_layout.process_shuffle_ram_init) {
      auto memory_cols_next_row = memory_cols.copy();
      if (gid < n - 1)
        memory_cols_next_row.add_row(1);
      else
        memory_cols_next_row.sub_row(gid);

      const bf intermediate_borrow = witness_cols.get_at_col(lazy_init_teardown_layout.init_address_intermediate_borrow);
      {
        const bf this_low = memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start);
        const bf next_low = memory_cols_next_row.get_at_col(lazy_init_teardown_layout.init_address_start);
        const bf aux_low = witness_cols.get_at_col(lazy_init_teardown_layout.init_address_aux_low);
        bf tmp = bf::mul(SHIFT_16, intermediate_borrow);
        tmp = bf::add(tmp, this_low);
        tmp = bf::sub(tmp, next_low);
        tmp = bf::sub(tmp, aux_low);
        const e4 alpha = (alphas_every_row_except_last_two++).get();
        acc_linear = e4::add(acc_linear, e4::mul(alpha, tmp));
      }
      {
        const bf final_borrow = witness_cols.get_at_col(lazy_init_teardown_layout.init_address_final_borrow);
        const bf this_high = memory_cols.get_at_col(lazy_init_teardown_layout.init_address_start + 1);
        const bf next_high = memory_cols_next_row.get_at_col(lazy_init_teardown_layout.init_address_start + 1);
        const bf aux_high = witness_cols.get_at_col(lazy_init_teardown_layout.init_address_aux_high);
        bf tmp = bf::mul(SHIFT_16, final_borrow);
        tmp = bf::add(tmp, this_high);
        tmp = bf::sub(tmp, intermediate_borrow);
        tmp = bf::sub(tmp, next_high);
        tmp = bf::sub(tmp, aux_high);
        const e4 alpha = (alphas_every_row_except_last_two++).get();
        acc_linear = e4::add(acc_linear, e4::mul(alpha, tmp));
      }
    }

    // Finalize "every row except last two" contributions, which are purely linear
    acc_linear = e4::mul(acc_linear, decompression_factor);
    multiplier = e2::mul(multiplier, e2::sub(x, omega_inv_squared));
    acc_linear = e4::mul(acc_linear, multiplier);
    acc = e4::add(acc, e4::mul(betas.get(4), acc_linear));
  }

  const e2 denoms[4] = {x, e2::sub(x, bf::one()), e2::sub(x, omega_inv_squared), e2::sub(x, omega_inv)};
  e2 denom_invs[4] = {};
  batch_inv_registers<e2, 4, true>(denoms, denom_invs, 4);

  // Constraints at first row: grand product == 1, boundary constraints
  {
    e4 acc_linear = e4::mul((helpers++).get(), stage_2_e4_cols.get_at_col(memory_grand_product_col));
    unsigned i = 0;
    if (lazy_init_teardown_layout.process_shuffle_ram_init)
      for (; i < 6; i++)
        acc_linear = e4::add(acc_linear, e4::mul((helpers++).get(), memory_cols.get_at_col(boundary_constraints.first_row_cols[i])));
    for (; i < boundary_constraints.num_first_row; i++)
      acc_linear = e4::add(acc_linear, e4::mul((helpers++).get(), witness_cols.get_at_col(boundary_constraints.first_row_cols[i])));
    acc_linear = e4::add(acc_linear, constants_times_challenges->first_row);
    acc_linear = e4::mul(acc_linear, denom_invs[1]);
    acc = e4::add(acc, acc_linear);
  }

  // Boundary constraints at one before last row
  if (boundary_constraints.num_one_before_last_row > 0) {
    e4 acc_linear{};
    unsigned i = 0;
    if (lazy_init_teardown_layout.process_shuffle_ram_init) {
      acc_linear = e4::mul((helpers++).get(), memory_cols.get_at_col(boundary_constraints.one_before_last_row_cols[0]));
      i++;
      for (; i < 6; i++)
        acc_linear = e4::add(acc_linear, e4::mul((helpers++).get(), memory_cols.get_at_col(boundary_constraints.one_before_last_row_cols[i])));
    } else {
      acc_linear = e4::mul((helpers++).get(), witness_cols.get_at_col(boundary_constraints.one_before_last_row_cols[0]));
      i++;
    }
    for (; i < boundary_constraints.num_one_before_last_row; i++)
      acc_linear = e4::add(acc_linear, e4::mul((helpers++).get(), witness_cols.get_at_col(boundary_constraints.one_before_last_row_cols[i])));
    acc_linear = e4::add(acc_linear, constants_times_challenges->one_before_last_row);
    acc_linear = e4::mul(acc_linear, denom_invs[2]);
    acc = e4::add(acc, acc_linear);
  }

  // One constraint at last row (grand product accumulator)
  {
    e4 acc_linear = e4::mul((helpers++).get(), stage_2_e4_cols.get_at_col(memory_grand_product_col));
    acc_linear = e4::add(acc_linear, (helpers++).get());
    acc_linear = e4::mul(acc_linear, denom_invs[3]);
    acc = e4::add(acc, acc_linear);
  }

  // Constraints at last row and x = 0
  {
    e4 acc_linear = e4::neg(stage_2_e4_cols.get_at_col(range_check_16_multiplicities_layout.dst_cols_start));
    // validate col sums for range check 16 lookup e4 args
    {
      const unsigned num_range_check_16_e4_args = range_check_16_layout.num_dst_cols + expressions.num_range_check_16_expression_pairs;
      for (unsigned i = 0; i < num_range_check_16_e4_args; i++)
        acc_linear = e4::add(acc_linear, stage_2_e4_cols.get_at_col(range_check_16_layout.e4_args_start + i));
      if (lazy_init_teardown_layout.process_shuffle_ram_init)
        acc_linear = e4::add(acc_linear, stage_2_e4_cols.get_at_col(lazy_init_teardown_layout.e4_arg_col));
      acc_linear = e4::mul(acc_linear, (helpers++).get());
    }
    // validate col sums for timestamp range check e4 args
    if (timestamp_range_check_multiplicities_layout.num_dst_cols > 0) {
      e4 acc_timestamp = e4::neg(stage_2_e4_cols.get_at_col(timestamp_range_check_multiplicities_layout.dst_cols_start));
      const unsigned num_timestamp_e4_args = expressions.num_timestamp_expression_pairs + expressions_for_shuffle_ram.num_expression_pairs;
      // This start location and the contiguity of e4 args cols are checked on the Rust side.
      const unsigned start_e4_col = (expressions.num_timestamp_expression_pairs > 0) ? expressions.e4_dst_cols[expressions.num_range_check_16_expression_pairs]
                                                                                     : expressions_for_shuffle_ram.e4_dst_cols[0];
      for (unsigned i = 0; i < num_timestamp_e4_args; i++)
        acc_timestamp = e4::add(acc_timestamp, stage_2_e4_cols.get_at_col(start_e4_col + i));
      acc_timestamp = e4::mul(acc_timestamp, (helpers++).get());
      acc_linear = e4::add(acc_linear, acc_timestamp);
    }
    // validate col sums for generic lookup e4 args
    {
      e4 acc_generic = e4::neg(stage_2_e4_cols.get_at_col(generic_lookup_multiplicities_layout.dst_cols_start));
      for (unsigned i = 1; i < generic_lookup_multiplicities_layout.num_dst_cols; i++)
        acc_generic = e4::sub(acc_generic, stage_2_e4_cols.get_at_col(generic_lookup_multiplicities_layout.dst_cols_start + i));
      for (unsigned i = 0; i < width_3_lookups_layout.num_lookups; i++)
        acc_generic = e4::add(acc_generic, stage_2_e4_cols.get_at_col(width_3_lookups_layout.e4_arg_cols_start + i));
      acc_generic = e4::mul(acc_generic, (helpers++).get());
      acc_linear = e4::add(acc_linear, acc_generic);
    }
    if (handle_delegation_requests || process_delegations) {
      const e4 interpolant = e4::mul((helpers++).get(), x);
      const e4 e4_arg = stage_2_e4_cols.get_at_col(delegation_aux_poly_col);
      const e4 diff = e4::sub(e4_arg, interpolant);
      const e4 term = e4::mul(diff, (helpers++).get());
      acc_linear = e4::add(acc_linear, term);
    }
    const e2 denom_inv = e2::mul(denom_invs[0], denom_invs[3]);
    acc_linear = e4::mul(acc_linear, denom_inv);
    acc = e4::add(acc, acc_linear);
  }

  quotient.set(acc);
}
